/*
 1.Input Data
 2.What Need to be calculated
 3.Design your threads and thread blocks
 4. Implementation on CPU and GPU
 5. Built in check points
 6. Output data
 */

 #include<stdio.h>
 #include<stdlib.h>
 #include<string.h>
 #include<math.h>
 #include<hip/hip_runtime.h>
 
 #include<time.h>

 #define NumberOfELements 100000
 #define PI 3.14159265

#define CHECK(call) \
{ \
 const hipError_t error = call; \
 if (error != hipSuccess) \
 { \
 printf("Error: %s:%d, ", __FILE__, __LINE__); \
 printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
 exit(1); \
 } \
}


int line_length(FILE *input_file){
    char read_lines[100];
    int total_lines = 0;
    while(fgets(read_lines, 100, input_file) != NULL) total_lines++;
        rewind(input_file);
      return(total_lines);
}

void get_data(FILE *input_file, int n_lines, float *asc, float *decl){
     char read_lines[100];
     float right_asc, declin;
     int i=0;
      while(fgets(read_lines, 100, input_file) != NULL){
       sscanf(read_lines, "%f  %f", &right_asc, &declin);
               asc[i] = right_asc * PI/ (60 * 180);
               decl[i] = declin * PI/ (60 * 180);
               ++i;
       }

    fclose(input_file);
  }

  __global__ void histogram_calc(float *rt_rl, float *decn_rl,  float *rt_syc, float *decn_syc, float pi, unsigned long long int *histogram){
           float galxs_rdns;
           float galxs_dgrs;
           int index  = blockIdx.x * blockDim.x + threadIdx.x;

         if( index < NumberOfELements)
            for( int i = 0; i < NumberOfELements; ++i){
               galxs_rdns =  acos(sin(decn_rl[index]) * sin(decn_syc[i]) + cos(decn_rl[index]) * cos(decn_syc[i]) * cos(rt_rl[index] - rt_syc[i]));
               galxs_dgrs = galxs_rdns * (180 /pi);
             // histogram[(int)(galxs_dgrs*4)] = (histogram[(int)(galxs_dgrs*4)] + 1);
                atomicAdd(&histogram[(int)(galxs_dgrs*4)], 1);
               __syncthreads();
          }
}

int  main(int argc, char *argv[]) {
     FILE *input_file, *output_file;
     unsigned long long int *DD, *DR, *RR;
     int total_lines_r, total_lines_s;
     float *right_ascension_real, *declination_real, *right_ascension_synthetic, *declination_synthetic;
     long int sum_DD, sum_DR, sum_RR;
     float *d_DC, *d_DR, *d_RR, *d_RC;
     double omg = 0.00;
     int bin_width = 4;
     int degrees = 180;
     int num_of_bins =
        num_of_bins =  bin_width * degrees;

     time_t start, stop;


      /* Check that we have 4 command line arguments */
       if ( argc != 4 ) {
          printf("Usage: %s real_data synthetic_data output_file\n", argv[0]);
          return(0);
         }

        start = clock();

        //open real data file
         input_file = fopen(argv[1], "r");
         if (input_file == NULL){
                printf("file does not exist%s\n", argv[1]);
                return 0;
         }

        // count lines in a real file
        total_lines_r = line_length(input_file);
        //printf("%s contains %d lines\n", argv[1],  total_lines_r);

        //alocate memory for real data on host
         right_ascension_real = (float *)calloc(total_lines_r, sizeof(float));
         declination_real =  (float *)calloc(total_lines_r, sizeof(float));

         //get data
         get_data(input_file, total_lines_r, right_ascension_real, declination_real);
        //open synthetic data
        input_file = fopen(argv[2], "r");
        if (input_file == NULL){
              printf("file does not exist%s\n", argv[2]);
              return 0;
         }
         //count lines in sysnthetic file
        total_lines_s = line_length(input_file);

        // printf("%s contains %d lines\n", argv[2], total_lines_s);

         //alocate memory for the sysnthetic data on host
         right_ascension_synthetic = (float *)calloc(total_lines_s, sizeof(float));
         declination_synthetic = (float *)calloc(total_lines_s, sizeof(float));

         //get second data
        get_data(input_file, total_lines_s,right_ascension_synthetic, declination_synthetic);

        // where data is stored
        long int *host_DD;
        long int *host_DR;
        long int *host_RR;

        //Alocate memory for the host
         host_DD = (long int *)malloc((num_of_bins+1)  * sizeof(long int));
         host_DR = (long int *)malloc((num_of_bins+1) * sizeof(long int));
         host_RR = (long int *)malloc((num_of_bins+1) * sizeof(long int));
         for (int i = 0; i <= num_of_bins; ++i ) {
              host_DD[i] = 0L;
              host_DR[i] = 0L;
              host_RR[i] = 0L;
            }

        //Allocate device memory
        hipMalloc((void **)&DD, (NumberOfELements+1)  * sizeof(unsigned long long int));
        hipMalloc((void **)&DR, (NumberOfELements+1)  * sizeof(unsigned long long int));
        hipMalloc((void **)&RR, (NumberOfELements+1)  * sizeof(unsigned long long int));

        hipMalloc((void **)&d_DR, (NumberOfELements+1) * sizeof(float));
        hipMalloc((void **)&d_DC, (NumberOfELements+1) * sizeof(float));
        hipMalloc((void **)&d_RR, (NumberOfELements+1) * sizeof(float));
        hipMalloc((void **)&d_RC, (NumberOfELements+1) * sizeof(float));

         //copy the data from host memory  to device memory
         hipMemcpy(d_DR, right_ascension_real, (NumberOfELements) * sizeof(float), hipMemcpyHostToDevice);
         hipMemcpy(d_DC, declination_real, (NumberOfELements) * sizeof(float), hipMemcpyHostToDevice);
         hipMemcpy(d_RR, right_ascension_synthetic, (NumberOfELements) * sizeof(float), hipMemcpyHostToDevice);
         hipMemcpy(d_RC, declination_synthetic, (NumberOfELements) * sizeof(float), hipMemcpyHostToDevice);


         //Lauch  the kernel for DD
          int blockSize = 256;
          int numBlocks = ((NumberOfELements -1) + blockSize - 1) / blockSize;
          histogram_calc <<<numBlocks, blockSize>>>(d_DR, d_DC, d_DR, d_DC,PI, DD);
          hipDeviceSynchronize();
          //copy the results back to the host
          hipMemcpy(host_DD, DD, num_of_bins * sizeof(long int), hipMemcpyDeviceToHost);

          sum_DD = 0L;
          for (int i = 0; i <= (num_of_bins); ++i )
                sum_DD += host_DD[i];
            printf("histograms DD = %ld\n", sum_DD);


         //Lauch  the kernel DR
          histogram_calc <<<numBlocks, blockSize>>>(d_DR, d_DC, d_RR, d_RC,PI, DR);
          hipDeviceSynchronize();
          //copy the results back to the host
          hipMemcpy(host_DR, DR, num_of_bins * sizeof(long int), hipMemcpyDeviceToHost);

          sum_DR = 0L;
          for (int i = 0; i <= num_of_bins; ++i )
                sum_DR += host_DR[i];
             printf("histograms DR = %ld\n", sum_DR);

          //Lauch  the kernel RR
          histogram_calc <<<numBlocks, blockSize>>>(d_RR, d_RC, d_RR, d_RC, PI, RR);
          //copy the results back to the host
          hipMemcpy(host_RR, RR, num_of_bins * sizeof(long int), hipMemcpyDeviceToHost);

          sum_RR = 0L;
          for (int i = 0; i <= num_of_bins; ++i )
                sum_RR += host_RR[i];
            printf("histograms RR = %ld\n", sum_RR);


        /* Open the output file */
        output_file = fopen(argv[3],"w");
        if ( output_file == NULL ) {
                printf("Unable to open %s\n",argv[3]);
                return(-1);
        }

       for(int i = 0; i < num_of_bins; ++i){
         if (host_RR[i] > 0 ) {
         omg = ((double)host_DD[i]/(double)(host_RR[i])) - ((2.0*host_DR[i])/(double)(host_RR[i])) + ((double)host_RR[i]/(double)(host_RR[i]));
        // omg = (double)((host_DD[i] - 2*host_DR[i] + host_RR[i])/host_RR[i]);
         printf("Omega = %6.3f\n", omg);
         fprintf(output_file, "%6.3f\n", omg);
          }
        }

        fclose(output_file);

        free(right_ascension_synthetic);
        free(declination_synthetic);
        free(right_ascension_real);
        free(declination_real);

        free(host_DD);
        free(host_DR);
        free(host_RR);

        hipFree(DD);
        hipFree(DR);
        hipFree(RR);

        hipFree(d_RR);
        hipFree(d_RC);
        hipFree(d_DR);
        hipFree(d_DC);

        stop = clock();
        printf("\nExcution time = %6.1f seconds\n",
        ((double) (stop-start))/ CLOCKS_PER_SEC);
        return (0);
}
